#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}

typedef struct Particle
{
	double* position_i;
	double* velocity_i;
	double* pos_best_i;
	double err_best_i = -1;
	double err_i = -1;
} Particle;


double RNG_RANDOM()
{
	return rand() / (double)RAND_MAX;
}

double RNG_UNIFORM(int a, int b)
{
	return (a + (b - a) * RNG_RANDOM());
}

__device__ double gpuRandomNumberUniform(hiprandState_t state) {
	hiprand_init(0, 0, 0, &state);
	return hiprand_uniform(&state);
}

const int num_dimensions = 2;

double BOUNDS_SPHERE[] = { -10, 10,-10, 10 };
double BOUNDS_ROSENBROCK[] = { -2048, 2048, -2048, 2048 };
double BOUNDS_RASTRINGIN[] = { -5.12, 5.12, -5.12, 5.12 };
double BOUNDS_SCHWEFEL[] = { -500, 500, -500, 500 };


__device__ double sphere(Particle* x)
{
	double total = 0.0;
	for (int i = 0; i <= num_dimensions; i++)
	{
		total += pow(x->position_i[i], 2);
	}
	return total;
}

__device__ double rosenbrock(Particle* x)
{
	double total = 0.0;
	for (int i = 0; i <= num_dimensions; i++)
	{
		total += 100 * (pow(x->position_i[i], 2) - pow(x->position_i[i + 1], 2)) + pow(1 - x->position_i[i], 2);
	}
	return total;
}

__device__ double rastrigin(Particle* x)
{
	double total = 0.0;
	for (int i = 0; i <= num_dimensions; i++)
	{
		total += (pow(x->position_i[i], 2) - (10 * cos(2 * 3.14159265359 * x->position_i[i])) + 10);
	}
	return total;
}

__device__ double schwefel(Particle* x)
{
	double total = 0.0;
	for (int i = 0; i <= num_dimensions; i++)
	{
		total += (x->position_i[i] * sin(sqrt(fabs(x->position_i[i]))));
	}
	return -total;
}

__global__ void update_gbest(Particle* swarm, double* pos_best_g, double* err_best_g) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (swarm[i].err_i < *err_best_g || *err_best_g == -1)
	{
		pos_best_g[0] = swarm[i].position_i[0];
		pos_best_g[1] = swarm[i].position_i[1];
		*err_best_g = swarm[i].err_i;
	}
}

__global__ void update_position_velocity(Particle* swarm, double* bounds, double* pos_best_g)
{

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	double w = 0.5;
	int c1 = 1;
	int c2 = 2;

	for (int j = 0; j < num_dimensions; j++)
	{
		swarm[i].position_i[j] = swarm[i].position_i[j] + swarm[i].velocity_i[j];

		if (swarm[i].position_i[j] > bounds[j * num_dimensions + 1])
		{
			swarm[i].position_i[j] = bounds[j * num_dimensions + 1];
		}

		if (swarm[i].position_i[j] < bounds[j * num_dimensions + 0])
		{
			swarm[i].position_i[j] = bounds[j * num_dimensions + 0];
		}

		hiprandState_t state1, state2;
		double r1 = gpuRandomNumberUniform(state1);
		double r2 = gpuRandomNumberUniform(state2);
		double vel_cognitive = c1 * r1 * (swarm[i].pos_best_i[j] - swarm[i].position_i[j]);
		double vel_social = c2 * r2 * (pos_best_g[j] - swarm[i].position_i[j]);
		swarm[i].velocity_i[j] = w * swarm[i].velocity_i[j] + vel_cognitive + vel_social;
	}
}



__global__ void calculate_fitness(Particle* swarm) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	double err = sphere(&swarm[i]);
	swarm[i].err_i = err;
}

__global__ void evaluate_update_pbest(Particle* swarm) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (swarm[i].err_i < swarm[i].err_best_i || swarm[i].err_i == -1)
	{
		swarm[i].pos_best_i = swarm[i].position_i;
		swarm[i].err_best_i = swarm[i].err_i;
	}
}

void alloc_memory(Particle* particle, int num_dimensions, double* x0)
{
	double* velocity_i = (double*)malloc(sizeof(double) * num_dimensions);
	double* position_i = (double*)malloc(sizeof(double) * num_dimensions);
	double* pos_best_i = (double*)malloc(sizeof(double) * num_dimensions);

	hipMalloc(&particle->position_i, sizeof(double) * num_dimensions);
	hipMalloc(&particle->velocity_i, sizeof(double) * num_dimensions);
	hipMalloc(&particle->pos_best_i, sizeof(double) * num_dimensions);

	for (int i = 0; i < num_dimensions; i++)
	{
		velocity_i[i] = RNG_UNIFORM(-1, 1);
		position_i[i] = x0[i];
		pos_best_i[i] = 0.0;
	}

	hipMemcpy(particle->velocity_i, velocity_i, sizeof(double) * num_dimensions, hipMemcpyHostToDevice);
	hipMemcpy(particle->position_i, velocity_i, sizeof(double) * num_dimensions, hipMemcpyHostToDevice);
	hipMemcpy(particle->pos_best_i, velocity_i, sizeof(double) * num_dimensions, hipMemcpyHostToDevice);
}

void initial_particles(const int& num_particle, const int& num_dimension, double  initial[2], Particle* swarm)
{
	for (int i = 0; i < num_particle; i++)
	{
		Particle particle, p;
		alloc_memory(&particle, num_dimension, initial);
		swarm[i] = particle;
	}
}


int main()
{
	double initial[] = { 5, 5 };
	const int num_particle = 4096;
	double h_BOUNDS[] = { -10,10,-10,10 };
	double* BOUNDS;

	double* pos_best_g;
	double* err_best_g;
	Particle* swarm;
	const int THREAD_PER_BLOCK = 128;
	const int BLOCKS = num_particle / THREAD_PER_BLOCK;
	const int MAX_ITER = 36;
	const int N = 4;

	hipMallocManaged(&pos_best_g, sizeof(double) * num_dimensions);
	hipMallocManaged(&err_best_g, sizeof(double));
	hipMallocManaged(&swarm, sizeof(Particle) * num_particle);

	hipMalloc(&BOUNDS, sizeof(double) * num_dimensions * num_dimensions);
	hipMemcpy(BOUNDS, h_BOUNDS, sizeof(double) * num_dimensions * num_dimensions, hipMemcpyHostToDevice);

	*err_best_g = -1;
	initial_particles(num_particle, num_dimensions, initial, swarm);

	float time = 0.0, gtime = 0.0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	for (int i = 0; i < MAX_ITER; i++) {
		calculate_fitness << <BLOCKS, THREAD_PER_BLOCK >> > (swarm);
		evaluate_update_pbest << <BLOCKS, THREAD_PER_BLOCK >> > (swarm);
		update_gbest << <BLOCKS, THREAD_PER_BLOCK >> > (swarm, pos_best_g, err_best_g);
		update_position_velocity << <BLOCKS, THREAD_PER_BLOCK >> > (swarm, BOUNDS, pos_best_g);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	printf("Final Solution: [x:%.20f, y:% .20f] error: % .20f\n", pos_best_g[0], pos_best_g[1], *err_best_g);
	printf("Time elapsed on CPU: %f ms.\n", time);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipFree(swarm);
	hipFree(pos_best_g);
	hipFree(swarm);
	hipFree(BOUNDS);
}
